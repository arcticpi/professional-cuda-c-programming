
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void HelloFromGPU()
{
    printf("Hello World from GPU!\n");
}

int main()
{
    printf("Hello World from CPU!\n");

    HelloFromGPU<<<1, 10>>>();
    hipDeviceReset();

    return 0;
}